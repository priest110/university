//#include <cstdlib.h>
//#include <iostream.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

hipEvent_t start, stop;

#define TIME_RESOLUTION 1000000

#define NUM_BLOCKS 10
#define NUM_THREAD_PER_BLOCK 10
#define TILE_SIZE 16


struct timeval t;
long long unsigned cpu_time;

long long unsigned tt = 0;

void startTime() 
{
        gettimeofday(&t, NULL);
        cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime() 
{
        gettimeofday(&t, NULL);
        long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

        final_time -= cpu_time;

        printf("%llu us\n", final_time);

        tt += final_time;

        cpu_time = 0;
}

//Do Lab2
// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) 
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

//Do Lab2
void stopKernelTime (void) 
{
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f us have elapsed for the CUDA execution\n", milliseconds);
}

__global__
void multiplication_kernel(float *a, float *b, float *c, int N)
{
    __shared__ float tile_a[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_b[TILE_SIZE][TILE_SIZE];

    int lin = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0;

    int x;

    int size = N*N;

    if(col < N && lin < N)
    {
        for(int sub = 0; sub < gridDim.x; sub++)
        {
            x = lin * N + sub * TILE_SIZE + threadIdx.x;

            tile_a[threadIdx.y][threadIdx.x] = x >= size ? 0 : a[x];

            x = (sub * TILE_SIZE + threadIdx.y) * N + col;

            tile_b[threadIdx.y][threadIdx.x] = x >= size ? 0 : b[x];
            
            __syncthreads();

            for(int k = 0; k < TILE_SIZE; k++)
                sum += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];

            __syncthreads();
        }

        c[lin * N + col] = sum;

    }
}

void fill_matrices(float **a, float **b, float **c, int N)
{
    int size = N * N;
    (*a) = (float*)malloc(sizeof(float) * size);
    (*b) = (float*)malloc(sizeof(float) * size);
    (*c) = (float*)malloc(sizeof(float) * size);

    srand(127);

    for(unsigned i = 0; i < size; i++)
    {
        (*b)[i] = 1;
        (*a)[i] = ((float)rand()/(float)RAND_MAX);
    }
}

void multiplication_stencil(float *a, float *b, float *c, int N)
{
    float *devA, *devB, *devC;
    int size = N * N;

    hipMalloc((float**) &devA, size * sizeof(float));
    hipMalloc((float**) &devB, size * sizeof(float));
    hipMalloc((float**) &devC, size * sizeof(float));

    startTime();

    hipMemcpy(devA, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, b, size * sizeof(float), hipMemcpyHostToDevice);

    stopTime();

    dim3 dimGrid(128,128);
    dim3 dimBlock(16,16);

    startKernelTime();

    multiplication_kernel<<<dimGrid,dimBlock>>>(devA, devB, devC, N);

    stopKernelTime();

    startTime();

    hipMemcpy(c, devC, size * sizeof(float), hipMemcpyDeviceToHost);

    stopTime();

    hipFree(devA);
    hipFree(devB);

    printf("Mem. Transfers took %lu us\n", tt);

    if(hipSuccess != hipGetLastError())
    {
        printf("Matrix Multiplication failed!");
        hipFree(devC);
        exit(-1);
    }
    hipFree(devC);
}

int main(int argc, char** argv)
{
    int N = 2048; //Largest data set size in 2.4
    float *a, *b, *c;
    for(int i = 0; i < 8; i++)
    {
        fill_matrices(&a,&b,&c,N);
        multiplication_stencil(a,b,c,N);
    }
}